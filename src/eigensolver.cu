#include <stdio.h>
#include <hip/hip_runtime.h>
#include <float.h>

__device__ void swap(float *a, float *b)
{
	float tmp = *a;
	*a = *b;
	*b = tmp;
}

extern __shared__ float BlockShMem[];
__global__ void CUDAJacobi(float *Aptr, float *Zptr, const unsigned int *Mptr, const int Nmat)
{
	const int N = sqrtf(Mptr[blockIdx.x]);
	float *A = Aptr + Mptr[blockIdx.x];
	float *Z = Zptr + Mptr[blockIdx.x];

	float *z = (float*)&BlockShMem[0];
	float *b = (float*)&BlockShMem[N];
	float *D = (float*)&BlockShMem[2 * N];

	// initialize Z as identity matrix
	//            D with A main diagonal
	//            b = D
	//            and z = 0
	for (int i = 0; i < N; ++i) {

		D[i] = A[i * N + i];
		b[i] = D[i];
		z[i] = 0.0;

		for (int j = 0; j < N; ++j) {
			Z[i * N + j] = (i == j)? 1.0 : 0.0;
		}
	}

	float g, h, t, theta, c, s, tau;
	const float EPSILON = FLT_EPSILON;
	int iteration = -1;
	const int MAX_ITERATIONS = 50;

	for (iteration = 0; iteration < MAX_ITERATIONS; ++iteration) {

		// convergence test: if we have a diagonal matrix (to machine precision) we're done
		float sm = 0.0;
		for (int i = 0; i < N - 1; ++i)
			for (int j = i + 1; j < N; ++j)
				sm += fabs(A[i * N + j]);

		if (sm == 0.0f)
			break;

		for (int ip = 0; ip < N - 1; ++ip) {
			for (int iq = ip + 1; iq < N; ++iq) {

				float Apq = A[ip * N + iq];
				g = 100.0f * fabsf(Apq);

				if (iteration > 4 && (g <= EPSILON * fabsf(D[ip])) && (g <= EPSILON * fabsf(D[iq]))) {

					A[ip * N + iq] = 0.0f;

				} else if (fabsf(Apq) > 0.0f) {

					h = D[iq] - D[ip];
					if (g <= EPSILON * fabsf(h)) {
						t = Apq / h;
					} else {
						theta = 0.5f * h / Apq;
						t = 1.0f / (fabsf(theta) + sqrtf(1.0f + theta * theta));
						if (theta < 0.0f)
							t = -t;
					}
					c = 1.0f / sqrtf(1 + t * t);
					s = t * c;
					tau = s / (1.0f + c);
					h = t * Apq;

					z[ip] -= h;
					D[ip] -= h;

					z[iq] += h;
					D[iq] += h;

					A[ip * N + iq] = 0.0f;

					// case of rotations 0 <= j < p
					for (int j = 0; j < ip; ++j) {
						float Aij = A[j * N + ip];
						float Akl = A[j * N + iq];
						A[j * N + ip] = Aij - s * (Akl + Aij * tau);
						A[j * N + iq] = Akl + s * (Aij - Akl * tau);
					}

					// case of rotations p <= j < q
					for (int j = ip + 1; j < iq; ++j) {
						float Aij = A[ip * N + j];
						float Akl = A[j * N + iq];
						A[ip * N + j] = Aij - s * (Akl + Aij * tau);
						A[j * N + iq] = Akl + s * (Aij - Akl * tau);
					}

					// case of rotations q <= j < n
					for (int j = iq + 1; j < N; ++j) {
						float Aij = A[ip * N + j];
						float Akl = A[iq * N + j];
						A[ip * N + j] = Aij - s * (Akl + Aij * tau);
						A[iq * N + j] = Akl + s * (Aij - Akl * tau);
					}

					// accumulate rotations for eigenvectors
					for (int j = 0; j < N; ++j) {
						float Aij = Z[j * N + ip];
						float Akl = Z[j * N + iq];
						Z[j * N + ip] = Aij - s * (Akl + Aij * tau);
						Z[j * N + iq] = Akl + s * (Aij - Akl * tau);
					}
				}
			}
		}

		for (int i = 0; i < N; ++i) {
			b[i] += z[i];
			D[i] = b[i];
			z[i] = 0.0;
		}
	}

	if (iteration < MAX_ITERATIONS) {

		// converged, now put eigenvalues in ascending order
		int min_idx;
		for (int i = 0; i < N; ++i) {
			min_idx = i;
			for (int j = i + 1; j < N; ++j)
				if (D[j] < D[min_idx])
					min_idx = j;

			if (i != min_idx)
				swap(&D[i], &D[min_idx]);
		}

	} else {

		for (int i = 0; i < N; ++i)
			D[i] = -1.0f;

	}
}


extern "C"
void eigenpairs_gpu(float *gpuA, float *gpuZ, const unsigned MaxMatrixSize, const unsigned *gpuMatrixSize, const int MatrixCount)
{
	dim3 gridDim(MatrixCount, 1);
	dim3 blockDim(1, 1);
	int shMemSize = 3 * MaxMatrixSize * sizeof(float);

	CUDAJacobi<<<gridDim, blockDim, shMemSize>>>(gpuA, gpuZ, gpuMatrixSize, MatrixCount);
}
